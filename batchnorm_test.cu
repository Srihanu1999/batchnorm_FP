#include <iostream>
#include <hip/hip_runtime.h>
#include "batchnorm_layer.h"

#define CHECK_CUDA(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess)
        std::cerr << "CUDA Error: " << hipGetErrorString(code) << " " << file << ":" << line << std::endl;
}

void print_tensor(const char* label, float* data, int N, int C, int H, int W) {
    std::cout << label << ":\n";
    for (int n = 0; n < N; ++n)
        for (int c = 0; c < C; ++c) {
            std::cout << "N=" << n << ", C=" << c << ":\n";
            for (int h = 0; h < H; ++h) {
                for (int w = 0; w < W; ++w) {
                    int idx = ((n * C + c) * H + h) * W + w;
                    std::cout << data[idx] << " ";
                }
                std::cout << "\n";
            }
        }
    std::cout << std::endl;
}

int main() {
    const int N = 1, C = 2, H = 2, W = 2;
    const int total = N * C * H * W;
    const float epsilon = 1e-5f;

    float h_input[total] = {1, 2, 3, 4, 5, 6, 7, 8};
    float h_gamma[C] = {1.0f, 1.0f};
    float h_beta[C] = {0.0f, 0.0f};

    float *d_input, *d_output, *d_mean, *d_var, *d_gamma, *d_beta;
    CHECK_CUDA(hipMalloc(&d_input, total * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, total * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_mean, C * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_var, C * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_gamma, C * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_beta, C * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_input, h_input, total * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_gamma, h_gamma, C * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_beta, h_beta, C * sizeof(float), hipMemcpyHostToDevice));

    batchnorm_forward(d_input, d_output, d_mean, d_var, d_gamma, d_beta, N, C, H, W, epsilon);

    float h_output[total], h_mean[C], h_var[C];
    CHECK_CUDA(hipMemcpy(h_output, d_output, total * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_mean, d_mean, C * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_var, d_var, C * sizeof(float), hipMemcpyDeviceToHost));

    print_tensor("Input Normalized", h_output, N, C, H, W);

    std::cout << "Mean: ";
    for (int i = 0; i < C; ++i) std::cout << h_mean[i] << " ";
    std::cout << "\nVariance: ";
    for (int i = 0; i < C; ++i) std::cout << h_var[i] << " ";
    std::cout << std::endl;

    hipFree(d_input); hipFree(d_output); hipFree(d_mean);
    hipFree(d_var); hipFree(d_gamma); hipFree(d_beta);
    return 0;
}
